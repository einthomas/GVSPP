#include "hip/hip_runtime.h"
#include "CUDAUtil.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/unique.h>
#include <thrust/device_ptr.h>
#include <thrust/tuple.h>
#include <thrust/functional.h>
#include <thrust/set_operations.h>
#include <thrust/binary_search.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/remove.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <chrono>
#include <vector>

#include "sample.h"
#include "gpuHashTable/linearprobing.h"

void sortByKey(thrust::device_ptr<int> triangleIDs, int size, thrust::device_ptr<int> sampleIndices);
int uniqueByKey(thrust::device_ptr<int> triangleIDs, int size, thrust::device_ptr<int> sampleIndices);
void findNewTriangles(
    thrust::device_ptr<int> devicePointerPVS, thrust::device_ptr<int> triangleIDs, thrust::device_ptr<Sample> samples, std::vector<Sample> &result, int pvsSize,
    int trianglesSize, thrust::device_ptr<int> sampleIndices
);
int setUnion(thrust::device_ptr<int> devicePointerPVS, thrust::device_ptr<int> triangleIDs, int sizeA, int sizeB);

struct is_even
{
  __host__ __device__
  bool operator()(const char x)
  {
    //return (x % 2) == 0;
      return x == 0;
  }
};

int CUDAUtil::work2(
    int *hashTable,
    int *pvs, int *triangleIDKeys, Sample *sampleValues, std::vector<Sample> &result, int pvsSize,
    const int triangleIDKeysSize
) {
    /*
    std::vector<int> sampleIndices = std::make_index_sequence<triangleIDKeysSize>{};
    uint32_t* deviceSampleIndices;
    hipMalloc(&deviceSampleIndices, sizeof(uint32_t) * sampleIndices.size());
    hipMemcpy(deviceSampleIndices, sampleIndices, sizeof(uint32_t) * sampleIndices.size(), hipMemcpyHostToDevice);
    */

    std::vector<char> inserted(triangleIDKeysSize);
    //std::cout << "insert a" << std::endl;
    insert_hashtable(hashTable, triangleIDKeys, triangleIDKeysSize, inserted.data());
    //std::cout << "insert b" << std::endl;

    thrust::device_vector<char> deviceInserted(inserted);

    int numNewTriangles = thrust::count(deviceInserted.begin(), deviceInserted.end(), 1);
    if (numNewTriangles > 0) {
        result.resize(numNewTriangles);

        thrust::device_ptr<Sample> devicePointerSampleValues(sampleValues);

        /*
        thrust::device_vector<Sample> r(numNewTriangles);
        thrust::copy_if(devicePointerSampleValues, devicePointerSampleValues + triangleIDKeysSize, deviceInserted.begin(), r.begin(), is_even());
        thrust::copy(r.begin(), r.end(), result.begin());
        */

        auto newEnd = thrust::remove_if(devicePointerSampleValues, devicePointerSampleValues + triangleIDKeysSize, deviceInserted.begin(), is_even()); //thrust::identity<char>());
        thrust::copy(devicePointerSampleValues, newEnd, result.begin());

        pvsSize += (newEnd - devicePointerSampleValues);

        /*
        std::cout << numNewTriangles << " " << (newEnd - devicePointerSampleValues) << " " << pvsSize << std::endl;

        std::cout << "pvssize " << pvsSize << std::endl;
        std::vector<int> aa(16);
        //int *pvsArray = new int[pvsSize];
        hipMemcpy(aa.data(), hashTable, sizeof(int) * 16, hipMemcpyDeviceToHost);
        for (int i = 0; i < 16; i++) {
            std::cout << "pvs " << aa[i] << std::endl;
        }
        for (int i = 0; i < result.size(); i++) {
            std::cout << result[i] << std::endl;
        }

        std::cout << std::endl << std::endl;
        */
    }


    /*
    // Count the number of triangles that are not in the PVS
    thrust::device_vector<char> deviceInserted(inserted);
    int numNewTriangles = thrust::count(deviceInserted.begin(), deviceInserted.end(), 1);
    hipDeviceSynchronize();

    if (numNewTriangles > 0) {
        // Remove the indices referring to samples that are already in the PVS
        thrust::remove_if(devicePointerSampleValueIndices, devicePointerSampleValueIndices + trianglesSize, stencil.begin(), thrust::identity<int>());
        hipDeviceSynchronize();

        // Store the new samples in a result vector
        thrust::device_vector<Sample> r(numNewTriangles);
        auto newEnd = thrust::gather(
            devicePointerSampleValueIndices, devicePointerSampleValueIndices + numNewTriangles,
            samples,
            r.begin()
        );
        hipDeviceSynchronize();

        result.resize(numNewTriangles);
        thrust::copy(r.begin(), r.end(), result.begin());
        hipDeviceSynchronize();
    }
    */

    return pvsSize;
}

int CUDAUtil::work(
    int *pvs, int *triangleIDKeys, Sample *sampleValues, std::vector<Sample> &result, int pvsSize,
    int triangleIDKeysSize
) {
    thrust::device_vector<int> deviceSampleValueIndices(triangleIDKeysSize);
    thrust::sequence(deviceSampleValueIndices.begin(), deviceSampleValueIndices.end());
    thrust::device_ptr<int> sampleIndices = deviceSampleValueIndices.data();
    thrust::device_ptr<int> devicePointerPVS(pvs);

    thrust::device_ptr<int> triangleIDs(triangleIDKeys);
    thrust::device_ptr<Sample> samples(sampleValues);

    sortByKey(triangleIDs, triangleIDKeysSize, sampleIndices);

    int numTriangles = uniqueByKey(triangleIDs, triangleIDKeysSize, sampleIndices);

    if (triangleIDs[0] == -1) {
        triangleIDs++;
        sampleIndices++;
        numTriangles--;
    }

    if (numTriangles > 0) {
        findNewTriangles(devicePointerPVS, triangleIDs, samples, result, pvsSize, numTriangles, sampleIndices);
        if (result.size() > 0) {
            pvsSize = setUnion(devicePointerPVS, triangleIDs, pvsSize, numTriangles);
        }
    }

    return pvsSize;
}

void sortByKey(thrust::device_ptr<int> triangleIDs, int size, thrust::device_ptr<int> sampleIndices) {
    thrust::sort_by_key(triangleIDs, triangleIDs + size, sampleIndices);
    hipDeviceSynchronize();
}

int uniqueByKey(thrust::device_ptr<int> triangleIDs, int size, thrust::device_ptr<int> sampleIndices) {
    auto newEnd = thrust::unique_by_key(triangleIDs, triangleIDs + size, sampleIndices);
    hipDeviceSynchronize();
    return newEnd.first - triangleIDs;
}

int setUnion(thrust::device_ptr<int> devicePointerPVS, thrust::device_ptr<int> triangleIDs, int sizeA, int sizeB) {
    thrust::device_vector<int> result(sizeA + sizeB);
    auto newEnd = thrust::set_union(
        devicePointerPVS, devicePointerPVS + sizeA, triangleIDs, triangleIDs + sizeB,
        result.begin(), thrust::less<int>()
    );
    hipDeviceSynchronize();

    int resultSize = newEnd - result.begin();
    thrust::copy(result.begin(), result.begin() + resultSize, devicePointerPVS);
    hipDeviceSynchronize();

    return resultSize;
}

void findNewTriangles(
    thrust::device_ptr<int> devicePointerPVS, thrust::device_ptr<int> triangleIDs, thrust::device_ptr<Sample> samples, std::vector<Sample> &result, int pvsSize,
    int trianglesSize, thrust::device_ptr<int> devicePointerSampleValueIndices
) {
    // Search which triangles are already in the PVS
    thrust::device_vector<bool> stencil(trianglesSize);
    thrust::binary_search(
        devicePointerPVS, devicePointerPVS + pvsSize,
        triangleIDs, triangleIDs + trianglesSize,
        stencil.begin()
    );
    hipDeviceSynchronize();

    // Count the number of triangles that are not in the PVS
    int numNewTriangles = thrust::count(stencil.begin(), stencil.end(), 0);
    hipDeviceSynchronize();

    if (numNewTriangles > 0) {
        // Remove the indices referring to samples that are already in the PVS
        thrust::remove_if(devicePointerSampleValueIndices, devicePointerSampleValueIndices + trianglesSize, stencil.begin(), thrust::identity<int>());
        hipDeviceSynchronize();

        // Store the new samples in a result vector
        thrust::device_vector<Sample> r(numNewTriangles);
        auto newEnd = thrust::gather(
            devicePointerSampleValueIndices, devicePointerSampleValueIndices + numNewTriangles,
            samples,
            r.begin()
        );
        hipDeviceSynchronize();

        result.resize(numNewTriangles);
        thrust::copy(r.begin(), r.end(), result.begin());
        hipDeviceSynchronize();
    }
}

__global__ void haltonKernel(int n, float *sequence, int startIndex) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int bases[4] = { 2, 3, 5, 7 };

    for (int i = 0; i < 4; i++) {
        float f = 1.0f;
        float r = 0.0f;
        int k = startIndex + offset + 1;
        while (k > 0) {
            f /= bases[i];
            r = r + f * (k % bases[i]);
            k = int(k / bases[i]);
        }
        sequence[offset * 4 + i] = r;
    }
}

void CUDAUtil::generateHaltonSequence(int n, float *sequence, int startIndex) {
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    haltonKernel<<<numBlocks, blockSize>>>(blockSize, sequence, startIndex);
    hipDeviceSynchronize();
}

// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
int CUDAUtil::initCuda(uint8_t *vkDeviceUUID, size_t UUID_SIZE) {
    int current_device = 0;
    int device_count = 0;
    int devices_prohibited = 0;

    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&device_count);

    if (device_count == 0) {
        fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    // Find the GPU which is selected by Vulkan
    while (current_device < device_count) {
        hipGetDeviceProperties(&deviceProp, current_device);

        if ((deviceProp.computeMode != hipComputeModeProhibited)) {
            // Compare the cuda device UUID with vulkan UUID
            int ret = memcmp((void*)&deviceProp.uuid, vkDeviceUUID, UUID_SIZE);
            if (ret == 0) {
                hipSetDevice(current_device);
                hipGetDeviceProperties(&deviceProp, current_device);
                printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
                 current_device, deviceProp.name, deviceProp.major,
                 deviceProp.minor);

                return current_device;
            }

        } else {
          devices_prohibited++;
        }

        current_device++;
    }

    if (devices_prohibited == device_count) {
        fprintf(stderr, "CUDA error: No Vulkan-CUDA Interop capable GPU found.\n");
        exit(EXIT_FAILURE);
    }

    return -1;
}
